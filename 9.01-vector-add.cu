
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<assert.h>

inline hipError_t checkCuda(hipError_t result){
    if(result != hipSuccess){
        fprintf(stderr,"CUDA Runtime Error :%s\n",hipGetErrorString(result));
        assert(result == hipSuccess)   ;    
    }
    return result;
}
void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int grid = gridDim.x * blockDim.x;
  
  for(int i = index; i < N; i+=grid)
  {
  if(i>=N)return;
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a, *b, *c;

  checkCuda(hipMallocManaged(&a,size));
  checkCuda(hipMallocManaged(&b,size));
  checkCuda(hipMallocManaged(&c,size));

  initWith(3, a, N);
  initWith(4, b, N);
  initWith(0, c, N);

  int per_block_size = 1024,per_grid_size = 32; 
  addVectorsInto<<<per_grid_size,per_block_size>>>(c, a, b, N);

  checkCuda( hipGetLastError() );
  
  checkCuda(hipDeviceSynchronize() );
  checkElementsAre(7, c, N);

  checkCuda(hipFree(a));
  checkCuda(hipFree(b));
  checkCuda(hipFree(c));
}