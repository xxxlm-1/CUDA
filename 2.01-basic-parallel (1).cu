
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  firstParallel<<<1,5>>>();
  hipDeviceSynchronize();
}
/*
This should be running in parallel.
This should be running in parallel.
This should be running in parallel.
This should be running in parallel.
This should be running in parallel.
*/