
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

 // for (int i = idx; i < N + stride; i += stride)
  for (int i = idx; i < N ; i += stride)
  {
    if(i>=N)return;
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

void checkError(hipError_t err){
    if(err !=hipSuccess)
    printf("ERR: %s \n",hipGetErrorString(err));
}
int main()
{

  int N = 10000;
  int *a;
  hipError_t err ;
  size_t size = N * sizeof(int);
  
  err = hipMallocManaged(&a, size);
  checkError(err);
 
  init(a, N);
  size_t threads_per_block = 1024;
    /* size_t threads_per_block = 2048; */
  size_t number_of_blocks = 32;
  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  err = hipGetLastError();
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
